#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define BLOCK_DIM 1024 // number of threads in a block

/* Here we do an inclusive scan of 'array' in place.
   'size' is the number of elements in 'array'.
   it should be a power of 2.
 
   We assume that 'array' is in shared memory so that there is no need to 
   copy it to shared memory here.
    */

__device__ void scan_plus(int *array, int size)
{
   for (unsigned int stride=1; stride <= size/2; stride *= 2) {
        int v;
        if (threadIdx.x >= stride) {
            v = array[threadIdx.x - stride];
        }
        __syncthreads(); /* wait until all threads finish reading 
		                    an element */

        if (threadIdx.x >= stride)
            array[threadIdx.x] += v;

        __syncthreads(); /* wait until all threads finish updating an
		                    element */
     }
     
} // scan_plus

/*
   This kernel compares the two strings s1 and s2. Both strings are
   terminated with a null byte.
   The result is an integer:  0, if s1 and s2 are equal;
                              a negative value if s1 is less than s2;
                              a positive value if s1 is greater than s2
   The argument 'result' is used to "return" the result.
   The arguments n1, n2 indicate the number of characters in s1 and s2, respectively
    (including the null byte at the end).
             
   We assume that the number of threads in a block is >= max(n1,n2).  
*/
__global__ void my_strcmp(const char  *s1, int n1, const char *s2, int n2,  int *result)
{
     __shared__ int flags[BLOCK_DIM];
     __shared__ int r;
     int tid = threadIdx.x;
    if(tid == 0) r = 0;
    __syncthreads();
    if (tid < n1 && tid <n2)
        flags[tid] = (s1[tid] - s2[tid]); 
    else
        flags[tid] =0; 
     __syncthreads();

     scan_plus(flags, BLOCK_DIM);
     __syncthreads();
     
    r = r +flags[tid]; 
    __syncthreads();
    *result = r;
    __syncthreads();

}


int main(int argc, char **argv) 
{

	char *dev_s1, *dev_s2;
    int *dev_result;
#if 0
    char s1[] = "supercalifragilisticexpialidocious";
    char s2[] = "supercalifragilisticexpialidocious";
#endif
    const char *s1, *s2; 

    if (argc == 3) {
        s1 = strdup(argv[1]);
        s2 = strdup(argv[2]);
    }
    else if (argc == 1) {
        /* read 2 strings from the standard input */
        if (scanf("%ms %ms", &s1, &s2) != 2) {
            fprintf(stderr, "invalid input\n");
            exit(1);
        }
    }
    else {
        fprintf(stderr, "usage: %s [<first string> <second string>]\n", argv[0]);
        exit(1);
    }
    /*
    nvcc -gencode arch=compute_61,code=sm_61 scan_strcmp.cu -o foo
cc1plus: fatal error: scan_strcmp.cu: No such file or directory
compilation terminated.
make: *** [makefile:2: finale] Error 1

    */

    int n1 = strlen(s1)+1; // null byte at the end is also counted
    int n2 = strlen(s2)+1;
           
    // allocate the memory on the GPU
    hipMalloc((void**)&dev_s1, n1);
    hipMalloc((void**)&dev_s2, n2);
    hipMalloc((void**)&dev_result, sizeof(int));
    
    hipMemcpy(dev_s1, s1, n1, hipMemcpyHostToDevice);
    hipMemcpy(dev_s2, s2, n2, hipMemcpyHostToDevice);
    
    int threadsPerBlock = BLOCK_DIM;
    int numOfBlocks = 1;
 
    my_strcmp<<<numOfBlocks, threadsPerBlock>>>(dev_s1, n1, dev_s2, n2, dev_result);
 
    // copy the result back from the GPU to the CPU
    int result;
    hipMemcpy(&result, dev_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("result is %d\n", result);
		
	    
    // free memory on the GPU side
    hipFree(dev_s1);
    hipFree(dev_s2);
    hipFree(dev_result);
    
}
